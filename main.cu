
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdlib>

// add two arrays
template<typename T>
__global__ void add(T *output, T *inputA, T *inputB) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main () {
  std::cout << "HEYO" << '\n';

  int N = 8000 * 8000; // 800px x 800px image
  int iterations = 10;
  int size = N*sizeof(float);
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  
  float *x, *y, *output;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, size);
  hipMallocManaged(&y, size);
  hipMallocManaged(&output, size);
  
  // initialize random arrays
  for (int i = 0; i < N; i++) {
    x[i] = ((float) std::rand()) / (float) RAND_MAX;
    y[i] = ((float) std::rand()) / (float) RAND_MAX;
  }

  for (int blerp = 0; blerp < iterations; blerp++) {
    add<<<numBlocks, blockSize>>>(output, x, y);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
  }

  // Free memory
  hipFree(x);
  hipFree(y);
  

  return 0;
}


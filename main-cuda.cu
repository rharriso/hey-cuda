#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>

// add two arrays
template<typename T>
__global__ void add(T *output, T *inputA, T *inputB) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  output[idx] = inputA[idx] + inputB[idx];
}

template<typename T>
__global__ void initRandom(T *arr, float minValue, float maxValue) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  thrust::default_random_engine rng;
  thrust::uniform_real_distribution<float> dist(minValue, maxValue);
  rng.discard(idx);
  arr[idx] = dist(rng);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main () {
  int N = 8000 * 8000; // 800px x 800px image
  int iterations = 10;
  int size = N*sizeof(float);
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  
  float *x, *y, *output;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, size);
  hipMallocManaged(&y, size);
  hipMallocManaged(&output, size);
   
  // initialize arrays 
  initRandom<<<numBlocks, blockSize>>>(x, 0., 1.);
  initRandom<<<numBlocks, blockSize>>>(y, 0., 1.);
  hipDeviceSynchronize();
  
  for (int blerp = 0; blerp < iterations; blerp++) {
    add<<<numBlocks, blockSize>>>(output, x, y);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
  }
  
  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(output);
  

  return 0;
}

